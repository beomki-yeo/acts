// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Acts/Plugins/Cuda/Utilities/CpuScalar.hpp"
#include "CudaUtils.cu"

namespace Acts{

template<typename Var_t>
class CudaScalar{

public:
  CudaScalar(){
    cudaErrChk( hipMalloc((Var_t**)&m_devPtr, sizeof(Var_t)) );
  }

  CudaScalar(Var_t* scalar){
    cudaErrChk( hipMalloc((Var_t**)&m_devPtr, sizeof(Var_t)) );
    cudaErrChk( hipMemcpy(m_devPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice) );
  }

  CudaScalar(const Var_t* scalar){
    cudaErrChk( hipMalloc((Var_t**)&m_devPtr, sizeof(Var_t)) );
    cudaErrChk( hipMemcpy(m_devPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice) );
  }
  
  ~CudaScalar(){ 
    cudaErrChk( hipFree(m_devPtr) );
  }

  Var_t* Get() { return m_devPtr; }

  Var_t GetHost() {
    Var_t* fHostPtr = new Var_t[1];
    cudaErrChk( hipMemcpy(fHostPtr, m_devPtr, sizeof(Var_t), hipMemcpyDeviceToHost) );
    return fHostPtr;
  }

  void Zeros() { hipMemset(m_devPtr,0,sizeof(Var_t)); }
  
  private:
  Var_t* m_devPtr;  
};
}
