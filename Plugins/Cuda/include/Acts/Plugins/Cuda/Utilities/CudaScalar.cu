// This file is part of the Acts project.
//
// Copyright (C) 2019 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Acts/Plugins/Cuda/Utilities/CpuScalar.hpp"
#include "CudaUtils.cu"

namespace Acts{

template<typename Var_t>
class CudaScalar{

public:
  CudaScalar(){
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t)) );
  }

  CudaScalar(Var_t* scalar){
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t)) );
    cudaErrChk( hipMemcpy(fDevPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice) );
  }

  CudaScalar(const Var_t* scalar){
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t)) );
    cudaErrChk( hipMemcpy(fDevPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice) );
  }
  
  ~CudaScalar(){ 
    cudaErrChk( hipFree(fDevPtr) );
  }

  Var_t* Get() { return fDevPtr; }

  Var_t GetHost() {
    Var_t* fHostPtr = new Var_t[1];
    cudaErrChk( hipMemcpy(fHostPtr, fDevPtr, sizeof(Var_t), hipMemcpyDeviceToHost) );
    return fHostPtr;
  }

  void Zeros() { hipMemset(fDevPtr,0,sizeof(Var_t)); }
  
  private:
  Var_t* fDevPtr;  
};
}
