// This file is part of the Acts project.
//
// Copyright (C) 2019 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Acts/Plugins/Cuda/Utilities/CpuVector.hpp"
#include "CudaUtils.cu"

namespace Acts{

template<typename Var_t>
class CudaVector{

public:
  
  CudaVector(size_t size){ 
    fSize = size;
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t)) );
  }

  CudaVector(size_t size, Var_t* vector){
    fSize = size;
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t)) );
    CopyH2D(vector, fSize, 0);
  }
     
  CudaVector(size_t size, Var_t* vector, size_t len, size_t offset){ 
    fSize = size;
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t)) );
    CopyH2D(vector, len, offset);
  }
  
  ~CudaVector(){ 
    hipFree(fDevPtr); 
  }

  size_t GetSize(){return fSize;}
  
  Var_t* Get(size_t offset=0) { return fDevPtr+offset; }

  Var_t* GetHost() {
    Var_t* fHostPtr = new Var_t[fSize];
    cudaErrChk( hipMemcpy(fHostPtr, fDevPtr, fSize*sizeof(Var_t), hipMemcpyDeviceToHost) );
    return fHostPtr;
  }

  void CopyH2D(Var_t* vector, size_t len, size_t offset){
    cudaErrChk( hipMemcpy(fDevPtr+offset, vector, len*sizeof(Var_t), hipMemcpyHostToDevice) );
  }
  void CopyH2D(Var_t* vector, size_t len, size_t offset, hipStream_t* stream){
    cudaErrChk( hipMemcpyAsync(fDevPtr+offset, vector, len*sizeof(Var_t), hipMemcpyHostToDevice, *stream) );
  }

  void Zeros(){
    cudaErrChk( hipMemset(fDevPtr, 0, fSize*sizeof(Var_t)) );
  }
  
private:
  Var_t* fDevPtr; 
  size_t fSize;
};
}
